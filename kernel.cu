#include "hip/hip_runtime.h"
﻿


using namespace std;

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------
char* aloca_sequencia(int n)
{
	char* seq;

	seq = (char*)malloc((n + 1) * sizeof(char));
	if (seq == NULL)
	{
		printf("\nErro na alocação de estruturas\n");
		exit(1);
	}
	return seq;
}
// ----------------------------------------------------------------------------
int** aloca_matriz(int n, int m)
{
	int** mat,
		i;

	mat = (int**)malloc((n + 1) * sizeof(int*));
	if (mat == NULL)
	{
		printf("\nErro na alocação de estruturas\n");
		exit(1);
	}
	for (i = 0; i <= n; i++)
	{
		mat[i] = (int*)malloc((m + 1) * sizeof(int));
		if (mat[i] == NULL)
		{
			printf("\nErro na alocação de estruturas\n");
			exit(1);
		}
	}
	return mat;
}
// ----------------------------------------------------------------------------
void distancia_edicao(int n, int m, char* s, char* r, int** d)
{
	int t, a, b, c, min, i, j;

	for (i = 1; i <= n; i++)
	{
		for (j = 1; j <= m; j++)
		{
			t = (s[i] != r[j] ? 1 : 0);
			a = d[i][j - 1] + 1;
			b = d[i - 1][j] + 1;
			c = d[i - 1][j - 1] + t;
			// Calcula d[i][j] = min(a, b, c)
			if (a < b)
				min = a;
			else
				min = b;
			if (c < min)
				min = c;
			d[i][j] = min;
		}
	}
}

//cuda
__global__ void calcula_adiagonal(int* tamMaxAdiag, int* indexLinhaInferior, char** r, char** s, int* indexColunaInferior, int* n, int* m, int*** d) 
{
	//calcular threadId e subtrair ele da linha e somar na coluna. assim, encontramos os index da matriz pelos quais cada thread é responsável
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	//check if there are more threads than elements to compute
	if(threadID > tamMaxAdiag[0]) {
		return;
	}

	int i = indexLinhaInferior[0] - threadId;
	int j = indexColunaInferior[0] + threadId;
	
	int t, a, b, c, min;

	t = (s[i] != r[j] ? 1 : 0);
	a = d[i][j - 1] + 1;
	b = d[i - 1][j] + 1;
	c = d[i - 1][j - 1] + t;

	// Calcula d[i][j] = min(a, b, c)
	if(a < b) {
		min = a;
	}else {
		min = b;
	}if (c < min) {
		min = c;
	}
	d[i][j] = min;
}

// ----------------------------------------------------------------------------
int distancia_edicao_adiagonal(int n, int m, char* s, char* r, int** d) //função alterada de void para int de forma a retornar a distancia
{
	int nADiag,			// Número de anti-diagonais
		tamMaxADiag,	// Tamanho máximo (número máximo de células) da anti-diagonal
		aD,				// Anti-diagonais numeradas de 2 a nADiag + 1
		k, i, j,
		t, a, b, c, min;

	nADiag = n + m - 1;
	tamMaxADiag = n;

	//Para cada anti-diagonal
	//lançar threads aqui, transformar o conteúdo desse for em uma função CUDA
	for (aD = 2; aD <= nADiag + 1; aD++)
	{
		//invocar kernel
	}
}

// ----------------------------------------------------------------------------
void libera(int n, char* s, char* r, int** d)
{
	int i;

	free(s);
	free(r);
	for (i = 0; i <= n; i++)
	{
		free(d[i]);
	}
}
// ----------------------------------------------------------------------------
int main(int argc, char** argv)
{
	int n,	// Tamanho da sequência s
		m,	// Tamanho da sequência r
		** d,	// Matriz de distâncias com tamanho (n+1)*(m+1)
		i, j;
	char* s,	// Sequência s de entrada (vetor com tamanho n+1)
		* r;	// Sequência r de entrada (vetor com tamanho m+1)
	FILE* arqEntrada;	// Arquivo texto de entrada

	if (argc != 2)
	{
		printf("O programa foi executado com argumentos incorretos.\n");
		printf("Uso: ./dist_seq <nome arquivo entrada>\n");
		exit(1);
	}

	// Abre arquivo de entrada
	arqEntrada = fopen(argv[1], "rt");

	if (arqEntrada == NULL)
	{
		printf("\nArquivo texto de entrada não encontrado\n");
		exit(1);
	}

	// Lê tamanho das sequências s e r
	fscanf(arqEntrada, "%d %d", &n, &m);

	// Aloca vetores s e r
	s = aloca_sequencia(n);
	r = aloca_sequencia(m);

	// Aloca matriz d
	d = aloca_matriz(n, m);

	// Lê sequências do arquivo de entrada
	s[0] = ' ';
	r[0] = ' ';
	fscanf(arqEntrada, "%s", &(s[1]));
	fscanf(arqEntrada, "%s", &(r[1]));

	// Fecha arquivo de entrada
	fclose(arqEntrada);

	//TODO: CUDA MALLOCS AQUI
	// Inicializa matriz de distâncias d
	for (i = 0; i <= n; i++)
	{
		d[i][0] = i;
	}
	for (j = 1; j <= m; j++)
	{
		d[0][j] = j;
	}

	
	// Calcula distância de edição entre sequências s e r
	//distancia_edicao(n, m, s, r, d);

	/* Paralelizar aqui
	ideia: jogar as matrizes e entradas para memoria compartilhada
	lançar threads de acordo com o numero de antidiagonais
	*/
	// Calcula distância de edição entre sequências s e r, por anti-diagonais
	int dist = distancia_edicao_adiagonal(n, m, s, r, d);


	printf("Distância=%d\n",dist);
	//printf("Tempo CPU = %.2fms\n", tempo);

	// Libera vetores s e r e matriz d
	libera(n, s, r, d);

	return 0;
}
// ----------------------------------------------------------------------------